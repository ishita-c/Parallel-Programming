
#include <hip/hip_runtime.h>
#include <map>
#include <string>
#include <vector>
#include <chrono>
#include <fstream>
#include <iostream>
#include <algorithm>

using namespace std;
using namespace std::chrono;

int readFourBytes(fstream& fs){
    unsigned char a[4];
    fs.read((char *)&a, sizeof(a));
    int x = (int)a[0] | (int)a[1]<<8 | (int)a[2]<<16 | (int)a[3]<<24; 
    return x;
}

int readTwoBytes(fstream& fs){
    unsigned char a[2];
    fs.read((char *)&a, sizeof(a));
    int x = (int)a[0] | (int)a[1]<<8;
    return x;
}

void writeFourBytes(uint32_t x, fstream& fs){
    char* buffer;
    buffer = (char*)(&x);

    for(int i = 0; i < 4; i++)
        fs.write(buffer + i, sizeof(char));
}

void readData(int m, int d, int k, int *blocks, int* indices, int *mat, string input_file){
    int b = m*m;
    fstream fs(input_file, ios::in | ios::binary);
    fs.seekg(12, std::ios::beg);
    vector<pair<int, vector<int>>> data(k);
    for(int i_ = 0 ; i_ < k ; i_++){
        int i = readFourBytes(fs);
        int j = readFourBytes(fs);
        vector<int> temp(m*m);
        for(int k_ = 0; k_ < b; k_++){
            int x = readTwoBytes(fs);
            // mat[i_*b + k_] = x;
            temp[k_] = x;
        }
        // blocks[i_] = i*d + j;
        data[i_] = {i*d + j, temp};
    }
    sort(data.begin(), data.end());
    map<int, int> mp;
    for(int i=0;i<k;i++){
        blocks[i] = data[i].first;
        if(mp[blocks[i]/d] == 0){
            mp[blocks[i]/d] = 1;
            indices[blocks[i]/d] = i;
        }
        for(int j=0;j<b;j++){
            mat[i*b + j] = data[i].second[j];
        }
    }

    fs.close();
}

//TODO: Complete this
__global__
void calculateResult(int n, int m, int k1, int k2, int* blocks1, int* indices1, int* mat1, int* blocks2, int* indices2, int* mat2, int* blocksOutput, uint32_t* matOutput){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int d = n/m;
    int b = m*m;
    if(idx >= d*d) return;

    int x = idx/d;
    int y = idx%d;
    uint32_t MAX_VAL = pow(2,32) - 1;

    for(int i=indices1[x];i<k1;i++){
        int x1 = blocks1[i]/d;
        if(x1!=x) break;
        int y1 = blocks1[i]%d;
        for(int j=indices2[y1];j<k2;j++){
            int x2 = blocks2[j]/d;
            if(x2!=y1) break;
            int y2 = blocks2[j]%d;
            if(y2!=y) continue;

            int offset1 = i*b;
            int offset2 = j*b;

            for(int i_ = 0;i_<m;i_++){
                for(int j_ = 0;j_<m;j_++){
                    long val = 0;
                    for(int k=0;k<m;k++){
                        val += mat1[offset1 + i_*m + k]*mat2[offset2 + k*m + j_];
                        if (val >= (long)MAX_VAL){
                            val = (long)MAX_VAL;
                            break;
                        } 
                    }
                    if(val > 0){
                        long new_val = matOutput[idx*b + i_*m + j_] + val;
                        if (new_val >= (long)MAX_VAL){
                            new_val = (long)MAX_VAL;
                        }
                        matOutput[idx*b + i_*m + j_] = (uint32_t)new_val;
                        blocksOutput[idx] = 1;
                    }
                }
            }
        }
    }
}

// Outputter
void outputResult(int n, int m, int* blocksOutput, uint32_t* matOutput, string output_file){
    int d = n/m;
    int k = 0;
    int b = m*m;
    for(int idx=0;idx<d*d;idx++){
        if(blocksOutput[idx]) k++;
    }
    fstream fs(output_file, ios::out | ios::binary);
    writeFourBytes(n, fs);
    writeFourBytes(m, fs);
    writeFourBytes(k, fs);

    cout << "Output k = " << k << endl;

    for(int idx=0;idx<d*d;idx++){
        if(blocksOutput[idx] == 0) continue;
        writeFourBytes(idx/d, fs);
        writeFourBytes(idx%d, fs);
        // cout << idx/d << " " << idx%d << endl;
        for(int i=0;i<m*m;i++){
            writeFourBytes(matOutput[idx*b + i], fs);
            // cout << matOutput[idx*b + i] << " " ;
        }
        // cout << endl;
    }
    fs.close();
}

int main(int argc, char* argv[]){

    auto start = high_resolution_clock::now();

    string input_file1 = argv[1];
    string input_file2 = argv[2];
    string output_file = argv[3];
    
    fstream fs1(input_file1, ios::in | ios::binary);
    int n1 = readFourBytes(fs1);
    int m1 = readFourBytes(fs1);
    int k1 = readFourBytes(fs1);
    fs1.close();

    fstream fs2(input_file2, ios::in | ios::binary);
    int n2 = readFourBytes(fs2);
    int m2 = readFourBytes(fs2);
    int k2 = readFourBytes(fs2);
    fs2.close();

    cout << "k1" << " = " << k1 << endl;
    cout << "k2" << " = " << k2 << endl;

    if(n1 != n2 || m1 != m2){
        cout << "Error: Matrices have different dimensions!" << endl;
    }

    int n = n1;
    int m = m1;

    cout << "n" << " = " << n << endl;
    cout << "m" << " = " << m << endl;
    // dimension of block matrix
    int d = n/m;

    vector<pair<int, vector<int>>> data1(k1), data2(k2);

    int *blocks1;
    int *blocks2;
    int *indices1;
    int *indices2;
    int *mat1;
    int *mat2;
    int* blocksOutput;
    uint32_t* matOutput;

    hipMallocManaged(&blocks1, k1*sizeof(int));
    hipMallocManaged(&blocks2, k2*sizeof(int));
    hipMallocManaged(&indices1, n*sizeof(int));
    hipMallocManaged(&indices2, n*sizeof(int));
    hipMallocManaged(&mat1, k1*m*m*sizeof(int));
    hipMallocManaged(&mat2, k2*m*m*sizeof(int));

    hipMallocManaged(&blocksOutput, d*d*sizeof(int));
    hipMallocManaged(&matOutput, n*n*sizeof(uint32_t));

    auto pre = high_resolution_clock::now();

    auto duration0 = duration_cast<microseconds>(pre - start);
    cout << "Time taken for initial allocation : "<< duration0.count()/(1000.0) << " ms" << endl;

    readData(m, d, k1, blocks1, indices1, mat1, input_file1);
    readData(m, d, k2, blocks2, indices2, mat2, input_file2);
    cout <<"Reading files done!" << endl;
    auto read = high_resolution_clock::now();

    auto duration1 = duration_cast<microseconds>(read - pre);
    cout << "Time taken for reading : "<< duration1.count()/(1000.0) << " ms" << endl;

    int num_threads_per_block = 1024;
    int num_blocks = d*d/num_threads_per_block + 1;

    cout << "Number of blocks: " << num_blocks << endl;
    cout << "Going to enter the kernel" <<endl;

    //Kernel function call
    calculateResult<<<num_blocks, num_threads_per_block>>>(n, m, k1, k2, blocks1, indices1, mat1, blocks2, indices2, mat2, blocksOutput, matOutput);

    hipError_t err = hipGetLastError();

    if ( err != hipSuccess )
    {
    printf("CUDA Error: %s\n", hipGetErrorString(err));       

    // Possibly: exit(-1) if program cannot continue....
    }

    hipDeviceSynchronize();

    auto process = high_resolution_clock::now();

    auto duration2 = duration_cast<microseconds>(process - read);
    cout << "Time taken for processing : "<< duration2.count()/(1000.0) << " ms" << endl;

    cout<<"Kernel job done !"<<endl;

    outputResult(n, m, blocksOutput, matOutput, output_file);

    auto end = high_resolution_clock::now();

    auto duration3 = duration_cast<microseconds>(end - process);
    cout << "Time taken for outputting : "<< duration3.count()/(1000.0) << " ms" << endl;

    auto duration4 = duration_cast<microseconds>(end - start);
    cout << "Total Time taken : "<< duration4.count()/(1000.0) << " ms" << endl;

    return 0;
}
